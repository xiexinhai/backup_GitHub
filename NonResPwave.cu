#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/resonances/NonResPwave.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>

namespace GooFit {

__device__ fpcomplex nonres_pwave(fptype m12, fptype m13, fptype m23, ParameterContainer &pc) {
    unsigned int cyclic_index = pc.getConstant(0);
    unsigned int spin         = 1;

//    fptype rMassSq    = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
//    fptype rMass = sqrt(rMassSq);
//    fptype mass_daug1 = PAIR_23 == cyclic_index ? c_daug2Mass : c_daug1Mass;
//    fptype mass_daug2 = PAIR_12 == cyclic_index ? c_daug2Mass : c_daug3Mass;
//    fptype mass_daug3 = PAIR_12 == cyclic_index ? c_daug3Mass : (PAIR_23 == cyclic_index?c_daug1Mass:c_daug2Mass);

    fpcomplex result{0.0, 0.0};
    fpcomplex ret(1.0, 0.0);
    ret *= spinFactor(spin, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass, m12, m13, m23, cyclic_index);

    result += ret;
    pc.incrementIndex(1, 0, 1, 0, 1);
    return result;
}

__device__ resonance_function_ptr ptr_to_NONRES_PWAVE = nonres_pwave;

namespace Resonances {

NonResPwave::NonResPwave(std::string name, Variable ar, Variable ai, unsigned int cyc)
    : ResonancePdf("NonResPwave", name, ar, ai) {
    registerConstant(cyc);
    registerFunction("ptr_to_NONRES_PWAVE", ptr_to_NONRES_PWAVE);
}

} // namespace Resonances
} // namespace GooFit
